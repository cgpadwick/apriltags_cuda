#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <memory>
#include <vector>
#include <NvInfer.h>
#include <hip/hip_runtime_api.h>
#include "cameraexception.h"
#include "opencv2/opencv.hpp"

// Logger for TensorRT info/warning/errors
class Logger : public nvinfer1::ILogger {
    void log(Severity severity, const char* msg) noexcept override {
        if (severity != Severity::kINFO) {
            std::cout << msg << std::endl;
        }
    }
} gLogger;

// Helper function to check CUDA errors
#define CHECK_CUDA(call) \
    do { \
        hipError_t status = call; \
        if (status != hipSuccess) { \
            std::cerr << "CUDA error at line " << __LINE__ << ": " \
                      << hipGetErrorString(status) << std::endl; \
            return false; \
        } \
    } while(0)

class TensorRTInference {
public:
    TensorRTInference(const std::string& enginePath) 
        : mEngine(nullptr), mContext(nullptr) {
        initializeEngine(enginePath);
    }

    ~TensorRTInference() {
        if (mContext) mContext->destroy();
        if (mEngine) mEngine->destroy();
        if (mRuntime) mRuntime->destroy();
        for (void* buf : mDeviceBuffers) {
            hipFree(buf);
        }
        if (mStream) hipStreamDestroy(mStream);
    }

    bool initializeEngine(const std::string& enginePath) {
        // Read engine file
        std::ifstream file(enginePath, std::ios::binary);
        if (!file.good()) {
            std::cerr << "Error opening engine file: " << enginePath << std::endl;
            return false;
        }

        file.seekg(0, std::ios::end);
        size_t size = file.tellg();
        file.seekg(0, std::ios::beg);

        std::vector<char> engineData(size);
        file.read(engineData.data(), size);
        file.close();

        // Create runtime and engine
        mRuntime = nvinfer1::createInferRuntime(gLogger);
        if (!mRuntime) {
            std::cerr << "Error creating TensorRT runtime" << std::endl;
            return false;
        }

        mEngine = mRuntime->deserializeCudaEngine(engineData.data(), size);
        if (!mEngine) {
            std::cerr << "Error deserializing CUDA engine" << std::endl;
            return false;
        }

        mContext = mEngine->createExecutionContext();
        if (!mContext) {
            std::cerr << "Error creating execution context" << std::endl;
            return false;
        }

        // Create CUDA stream
        CHECK_CUDA(hipStreamCreate(&mStream));

        // Allocate device buffers
        for (int i = 0; i < mEngine->getNbBindings(); i++) {
            nvinfer1::Dims dims = mEngine->getBindingDimensions(i);
            size_t size = 1;
            for (int j = 0; j < dims.nbDims; j++) {
                size *= dims.d[j];
            }
            size *= sizeof(float);  // Assuming float32 data type

            void* deviceBuffer;
            CHECK_CUDA(hipMalloc(&deviceBuffer, size));
            mDeviceBuffers.push_back(deviceBuffer);
            
            if (mEngine->bindingIsInput(i)) {
                mInputSize = size;
                mInputDims = dims;
            } else {
                mOutputSize = size;
                mOutputDims = dims;
            }
        }

        return true;
    }

    bool infer(const float* input, float* output) {
        // Copy input to device
        CHECK_CUDA(hipMemcpyAsync(
            mDeviceBuffers[0], input, mInputSize,
            hipMemcpyHostToDevice, mStream
        ));

        // Execute inference
        if (!mContext->enqueueV2(mDeviceBuffers.data(), mStream, nullptr)) {
            std::cerr << "Error running inference" << std::endl;
            return false;
        }

        // Copy output back to host
        CHECK_CUDA(hipMemcpyAsync(
            output, mDeviceBuffers[1], mOutputSize,
            hipMemcpyDeviceToHost, mStream
        ));

        // Synchronize stream
        CHECK_CUDA(hipStreamSynchronize(mStream));

        return true;
    }

    nvinfer1::Dims getInputDims() const { return mInputDims; }
    nvinfer1::Dims getOutputDims() const { return mOutputDims; }
    size_t getInputSize() const { return mInputSize; }
    size_t getOutputSize() const { return mOutputSize; }

private:
    nvinfer1::IRuntime* mRuntime;
    nvinfer1::ICudaEngine* mEngine;
    nvinfer1::IExecutionContext* mContext;
    hipStream_t mStream;
    std::vector<void*> mDeviceBuffers;
    size_t mInputSize;
    size_t mOutputSize;
    nvinfer1::Dims mInputDims;
    nvinfer1::Dims mOutputDims;
};

void printCameraSettings(const cv::VideoCapture& cap) {
    int frame_width = cap.get(cv::CAP_PROP_FRAME_WIDTH);
    int frame_height = cap.get(cv::CAP_PROP_FRAME_HEIGHT);
    int frame_rate = cap.get(cv::CAP_PROP_FPS);

    std::cout << "  " << frame_width << "x" << frame_height << " @"
              << frame_rate << "FPS" << std::endl;

    std::cout << " format is: " << cap.get(cv::CAP_PROP_FORMAT) << std::endl;

    std::cout << "AUTO Exposure: " << cap.get(cv::CAP_PROP_AUTO_EXPOSURE)
              << std::endl;
    std::cout << "Brightness: " << cap.get(cv::CAP_PROP_BRIGHTNESS)
              << std::endl;
    std::cout << "Contrast: " << cap.get(cv::CAP_PROP_CONTRAST) << std::endl;
  }

// Example usage
int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <engine_file>" << std::endl;
        return 1;
    }

    // Initialize inference engine
    TensorRTInference inference(argv[1]);

    // Get input dimensions
    auto inputDims = inference.getInputDims();
    size_t inputElements = 1;
    for (int i = 0; i < inputDims.nbDims; i++) {
        std::cout << inputDims.d[i] << std::endl;
        inputElements *= inputDims.d[i];
    }

    std::cout << inputDims.nbDims << ", " << inputElements << std::endl;

    // Prepare input data (example with random data)
    std::vector<float> input(inputElements);
    for (size_t i = 0; i < inputElements; i++) {
        input[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    cv::VideoCapture cap;
    bool camera_started = false;
    while (!camera_started) {
      try {
        cap.open(0, cv::CAP_V4L);
        if (cap.isOpened()) {
          camera_started = true;
          std::cout << "Camera started successfully on index " << 0
                    << std::endl;
        } else {
          throw CameraException();
        }
      } catch (const CameraException& e) {
        std::cout << "Couldn't open video capture device: " << e.what()
                  << std::endl;
        std::cout << "Retrying in 1 second ...";
      }
    }

    // Set video mode, resolution and frame rate.
    int fourcc = cv::VideoWriter::fourcc('M', 'J', 'P', 'G');
    cap.set(cv::CAP_PROP_FOURCC, fourcc);
    cap.set(cv::CAP_PROP_FRAME_WIDTH, 1280);
    cap.set(cv::CAP_PROP_FRAME_HEIGHT, 800);
    cap.set(cv::CAP_PROP_FPS, 30);
    cap.set(cv::CAP_PROP_CONVERT_RGB, true);

    printCameraSettings(cap);

    float mean[3] = {0.485f, 0.456f, 0.406f};       // ImageNet means
    float std[3]  = {0.229f, 0.224f, 0.225f};       // ImageNet std devs

    cv::Mat bgr_img, rgb_img, rgb_img_subsampled;
    int w = 640;
    int h = 640;
    while (1) {
        cap >> bgr_img;

        // Preprocess the data before putting it through the engine.
        cv::cvtColor(bgr_img, rgb_img, cv::COLOR_BGR2RGB);
        cv::resize(rgb_img, rgb_img_subsampled, cv::Size(h, w), 0, 0, cv::INTER_LINEAR);
        rgb_img_subsampled.convertTo(rgb_img_subsampled, CV_32F);
        rgb_img_subsampled /= 255.0f;

        // normalize by mean and std dev.
        std::vector<cv::Mat> channels;
        cv::split(rgb_img_subsampled, channels);
        for (int i=0; i < 3; ++i) {
            channels[i] = (channels[i] - mean[i]) / std[i];
        }

        std::vector<float> input_tensor(1 * 3 * h * w);
        float* input_buffer = input_tensor.data();
        for (int c=0; c<3; c++) {
            int channel_offset = c * h * w;
            float* channel_data = reinterpret_cast<float*>(channels[c].data);
            std::memcpy(input_buffer + channel_offset, 
                       channel_data, 
                       h * w * sizeof(float));
        }

        // Prepare output buffer
        std::cout << "outputsize: " << inference.getOutputSize() << std::endl;
        std::vector<float> output(inference.getOutputSize() / sizeof(float));

        // Run inference
        if (!inference.infer(input.data(), output.data())) {
            std::cerr << "Inference failed" << std::endl;
            return 1;
        }

        // Print first few outputs
        std::cout << "Output preview: ";
        for (int i = 0; i < std::min(5ul, output.size()); i++) {
            std::cout << output[i] << " ";
        }
        std::cout << std::endl;

    }

    

    
    

    return 0;
}
